#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce() {
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
	uint64_t nonce = 0; // TODO: Update
	size_t current_length;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);

	hipEvent_t start, stop;
	startTiming(&start, &stop);

	findNonce<<<1, 1>>>();

	float seconds = stopTiming(&start, &stop);
	printResult(block_hash, nonce, seconds);

	return 0;
}
